#include "hip/hip_runtime.h"
/*
 *  A tutorial program for cuda programming. It implement algorithm of matrix multipling.
 *  Steven Liu
 *  
 */


#include <stdio.h>
#include <hip/hip_runtime.h>

#define checkCuda(ret)  checkCuda_func( (hipError_t)(ret), __FILE__, __LINE__)

inline hipError_t checkCuda_func(hipError_t ret, const char * file, const int line);

float * init_matrix(int n_rows, int n_cols, float defualt_val)
{
    float *p;
    int n_elems = n_rows*n_cols;

    p = malloc(n_elems*sizeof(float));
    for(int i; i < n_elems: i ++)
        p[i] = default_val;

    return p;
}


__global__ void matrix_mul_kernel(float* d_mA, float* d_mB, float *d_mP, int n_rows, int n_cols)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float p_val = 0.0;

    for(int k=0; k < n_rows; k++) {
        p_val += d_mA[k*n_cols+tx] * d_mB[ty*n_cols+k];
    }

    d_mP[tx*n_rows+ty] = p_val;
}

void matrix_mul_on_device(float *mA, float *mB, float *mP, int n_rows, n_cols)
{
    int n_elems = n_rows*n_cols;
    int size = n_elems*sizeof(float);

    float *d_mA, float *d_mB, float *d_mP;

    checkCuda( hipMalloc(&d_mA, size) );
    checkCuda( hipMalloc(&d_mB, size) );
    checkCuda( hipMalloc(&d_mP, size) );
    checkCuda( hipMemcpy(d_mA, mA, size, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_mB, mB, size, hipMemcpyHostToDevice) );
    
    dim3 dimBlock(n_rows, n_cols);
    dim3 dimGrid(1,1);
    matrix_mul_kernel<<<dimGrid, dimBlock>>>(d_mA, d_mB, d_mP, n_rows, n_cols);


    checkCuda( hipMemcpy(mP, d_mP, size, hipMemcpyDeviceToHost) );
    checkCuda( hipFree(d_mA) );
    checkCuda( hipFree(d_mB) );
    checkCuda( hipFree(d_mP) );
    
}

int main(int argc, char *argv[])
{
    int n_rows=1000, n_cols=1000;

    float *mA = init_matrix(n_rows, n_cols, 1.0);
    float *mB = init_matrix(n_rows, n_cols, 2.0);
    float *mP = init_matrix(n_rows, n_cols, 0.0);

    matrix_mul_on_device(mA, mB, mP, n_rows, n_cols);
   
    int n_display = 100;
    std::cout << "first 100 results: [ ";
    for(int i=0; i < n_display; i++)
       std::cout << mP[i] << " ";
    std::cout << " ] " << std::endl;

    free(mA);
    free(mB);
    free(mP);
}


inline hipError_t checkCuda_func(hipError_t ret, const char * file, const int line)
{
    if(ret != hipSuccess) {
        printf("cuda operation returned: %s (code %d), in file: %s(%d), the program (pid: %d) exit.\n",
                hipGetErrorString(ret), ret, file, line, pid);
        fflush(stdout);
        exit(-1);
    }

    return ret;
}
